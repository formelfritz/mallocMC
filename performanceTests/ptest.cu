#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <hip/hip_runtime.h>


#ifndef MC_CONFIG_ID
  #include "mc_config1.cu"   // default config
  #define MC_CONFIG_ID 1
#else // choose config (set via cmake)
  #if MC_CONFIG_ID==1
    #include "mc_config1.cu"
  #elif MC_CONFIG_ID==2
    #include "mc_config2.cu"
  #elif MC_CONFIG_ID==3
    #include "mc_config3.cu"
  #endif
#endif



#define CHECK_CUDA(cmd) {hipError_t error = cmd; if(error!=hipSuccess){printf("<%s>:%i ",__FILE__,__LINE__); printf("[CUDA] Error: %s\n", hipGetErrorString(error));}}

/*only check if kernel start is valid*/
#define CHECK_CUDA_KERNEL(...) __VA_ARGS__;CHECK_CUDA(hipGetLastError())

// I know it is not a Object oriented style
const int arrCnt = 4;
const int minArrUsed = 2;
const int maxArrUsed = 4;

typedef char* (threadArr_t)[arrCnt];
__device__ threadArr_t * a;

uint64_t * clockTicks;
uint32_t * allocatedChunksCount;

ScatterAllocator *mmc = NULL;

// standard nvidia allocator
__global__ void cuNewCreateArrayPointer(int size){
  a = new threadArr_t[size];
}

__global__ void cuNewFreeArrayPointer(){
  delete [] a;
}

__global__ void cuNewArray(unsigned int chunkSize,
                           unsigned int maxChunkCountPerAlloc,
                           unsigned int arrIdx,
                           uint64_t *clockTicks,
                           uint32_t *allocCount){
  // allocate just some "random" requested size (in multipes of chunksize)
  unsigned int id = threadIdx.x + blockIdx.x*blockDim.x;
  unsigned int chunkAlloc = 
                  (1103515245 * arrIdx + id) % maxChunkCountPerAlloc + 1;
  unsigned int size = chunkSize*chunkAlloc;
  clock_t start = clock64();
  a[id][arrIdx] = new char[size];
  clock_t end = clock64();
  allocCount[id] = chunkAlloc;
  clockTicks[id] = end - start;
}

__global__ void cuFreeArray(unsigned int arrIdx){
   unsigned int id = threadIdx.x + blockIdx.x*blockDim.x;
   delete [] a[id][arrIdx];  
}


// mallocMC
__global__ void mmcCreateArrayPointer(int size, ScatterAllocator::AllocatorHandle mMC){
  a = (threadArr_t*) mMC.malloc(sizeof(threadArr_t*) * size);
}

__global__ void mmcNewFreeArrayPointer(ScatterAllocator::AllocatorHandle mMC){
  mMC.free(a);
}

__global__ void mmcNewArray(unsigned int chunkSize,
                           unsigned int maxChunkCountPerAlloc,
                           unsigned int arrIdx,
                           uint64_t *clockTicks,
                           uint32_t *allocCount,
                           ScatterAllocator::AllocatorHandle mMC){
  // allocate just some "random" requested size (in multipes of chunksize)
  unsigned int id = threadIdx.x + blockIdx.x*blockDim.x;
  unsigned int chunkAlloc = 
                  (1103515245 * arrIdx + id) % maxChunkCountPerAlloc + 1;
  unsigned int size = chunkSize*chunkAlloc;
  clock_t start = clock64();
  a[id][arrIdx] = (char*) mMC.malloc(size*sizeof(char));
  clock_t end = clock64();
  allocCount[id] = chunkAlloc;
  clockTicks[id] = end - start;
}

__global__ void mmcFreeArray(unsigned int arrIdx, ScatterAllocator::AllocatorHandle mMC){
   unsigned int id = threadIdx.x + blockIdx.x*blockDim.x;
   mMC.free(a[id][arrIdx]);
}

// host functions
int gpuInit(bool useMallocMC){
  int ccMajor = 0;
  hipDeviceGetAttribute(&ccMajor, hipDeviceAttributeComputeCapabilityMajor, 0);
  int ccMinor = 0;
  hipDeviceGetAttribute(&ccMinor, hipDeviceAttributeComputeCapabilityMinor, 0);

  if( ccMajor < 2 ) {
    printf("Error: Compute Capability >= 2.0 required. (is %d.%d)\n",
                  ccMajor, ccMinor);
    return -1;
  }
  CHECK_CUDA(hipSetDevice(0));
  size_t free, total;
  CHECK_CUDA(hipMemGetInfo(&free, &total));
  if(useMallocMC){
    size_t allocMem = (free *7) / 8;  // dont allocate/reserve all memory
    mmc = new ScatterAllocator(allocMem);
    return allocMem;
  }else{
    return free;
  }
}

void gpuFree(unsigned int arrIdx, int gs, int bs){
  if(mmc != NULL){
    CHECK_CUDA_KERNEL(mmcFreeArray<<<gs, bs>>>(arrIdx, *mmc));
  }else{
    CHECK_CUDA_KERNEL(cuFreeArray<<<gs, bs>>>(arrIdx));
  }
}

int gpuAlloc(unsigned int chunkSize, unsigned int maxChunkCountPerAlloc, 
                  unsigned int arrIdx, int gs, int bs, uint64_t *clockRes){
  if(mmc != NULL){
    CHECK_CUDA_KERNEL(mmcNewArray<<<gs, bs>>>(chunkSize,
                     maxChunkCountPerAlloc, arrIdx,
                     clockTicks, allocatedChunksCount, *mmc));
  }else{
    CHECK_CUDA_KERNEL(cuNewArray<<<gs, bs>>>(chunkSize,
                     maxChunkCountPerAlloc, arrIdx,
                     clockTicks, allocatedChunksCount));
  }
  int size = gs * bs;
  thrust::device_ptr<uint32_t> thAllocatedChunksCount =
            thrust::device_pointer_cast(allocatedChunksCount);
  //thrust::device_vector<uint64_t> ct(thClockTicks, thClockTicks+size);

  if(clockRes != NULL){
    thrust::device_ptr<uint64_t> thClockTicks = thrust::device_pointer_cast(clockTicks);
    uint64_t clockSum = thrust::reduce(thClockTicks, thClockTicks+size);
    uint64_t clockMin = *thrust::min_element(thClockTicks, thClockTicks + size);
    uint64_t clockMax = *thrust::max_element(thClockTicks, thClockTicks + size);

    clockRes[0] = clockMin;
    clockRes[1] = clockSum / size;
    clockRes[2] = clockMax;
  }
  return thrust::reduce(thAllocatedChunksCount, 
                        thAllocatedChunksCount + size);
}


void run(int chunkSize, int maxChunkCountPerAlloc, int warmUpRounds, int testRounds,
         int gridSize, int blockSize, ScatterAllocator *mmc, FILE *fp){        
  fprintf(fp, "%d %d %d %d %d %d %d %d %d %d\n", chunkSize, maxChunkCountPerAlloc,
               (mmc==NULL) ? 0 : 1, 
               warmUpRounds, testRounds, gridSize, blockSize, 
               arrCnt, minArrUsed, maxArrUsed); 
  // init round (request Memory)
  uint32_t size = blockSize * gridSize;
  CHECK_CUDA (hipMalloc(&clockTicks, sizeof(uint64_t) *size));
  CHECK_CUDA (hipMalloc(&allocatedChunksCount, sizeof(uint32_t) *size));
  if(mmc != NULL){
    CHECK_CUDA_KERNEL(mmcCreateArrayPointer<<<1,1>>>(size, *mmc));
  } else {
    CHECK_CUDA_KERNEL(cuNewCreateArrayPointer<<<1,1>>>(size));
  }

  // initial state of arrays
  int arrAllocated[arrCnt];  // stores sum of allocated chunks for each arrayNumber
  uint64_t clockTimingSum[arrCnt][3];
  uint64_t chunkRequestSum[arrCnt];
  uint64_t chunkAllocatedSum[arrCnt];
  int arrAllocRequestCount[arrCnt];
  for(int i=0; i<arrCnt; i++){
    arrAllocated[i] = 0;
    chunkRequestSum[i] = 0;
    chunkAllocatedSum[i] = 0;
    clockTimingSum[i][0] = 0;
    clockTimingSum[i][1] = 0;
    clockTimingSum[i][2] = 0;
    arrAllocRequestCount[i] = 0; 
  }

  // warm up 
  int arrAllocatedCnt = 0;
  for(int i=0; i<warmUpRounds; i++){
    int arrIdx = rand() % arrCnt;
    // update arrIdx, until alloc/free respects fill boundaries
    while( (arrAllocatedCnt<minArrUsed && arrAllocated[arrIdx]!=0 ) ||
           (arrAllocatedCnt>=maxArrUsed && arrAllocated[arrIdx]==0) ){
      arrIdx = rand() % arrCnt;
    }
    if(arrAllocated[arrIdx]){
      gpuFree(arrIdx, gridSize, blockSize);
      arrAllocated[arrIdx] = 0;
      arrAllocatedCnt--;
    } else {
      arrAllocated[arrIdx] = gpuAlloc(chunkSize, maxChunkCountPerAlloc,
                                     arrIdx, gridSize, blockSize,
                                     NULL);
      arrAllocatedCnt++;
    }
  }

  // repeat until number of allocation calls equals testRounds
  int validAllocCnt = 0;
  uint64_t clockTiming[3];
  while(validAllocCnt<testRounds){
    int arrIdx = rand() % arrCnt;
    // update arrIdx, until alloc/free respects fill boundaries
    while( (arrAllocatedCnt<=minArrUsed && arrAllocated[arrIdx]) ||
           (arrAllocatedCnt>=maxArrUsed && !arrAllocated[arrIdx])){
      arrIdx = rand() % arrCnt;
    }
    if(arrAllocated[arrIdx] != 0){
      gpuFree(arrIdx, gridSize, blockSize);
      arrAllocated[arrIdx] = 0;
      arrAllocatedCnt--;
    } else {
      int allocChunkSum = 0;
      for(int i=0; i<arrCnt; i++){
        allocChunkSum += arrAllocated[i];
      }
      arrAllocated[arrIdx] = gpuAlloc(chunkSize, maxChunkCountPerAlloc, 
                                    arrIdx, gridSize, blockSize,
                                    clockTiming);

      chunkAllocatedSum[arrAllocatedCnt] += allocChunkSum;
      chunkRequestSum[arrAllocatedCnt] += arrAllocated[arrIdx];
      clockTimingSum[arrAllocatedCnt][0] += clockTiming[0];
      clockTimingSum[arrAllocatedCnt][1] += clockTiming[1];
      clockTimingSum[arrAllocatedCnt][2] += clockTiming[2];
      arrAllocRequestCount[arrAllocatedCnt]++;
      // write single run infos to file
      /*fprintf(fp, "%d %d %d %s", arrAllocatedCnt, allocChunkSum,
                                 arrAllocated[arrIdx], clockStr); */
      arrAllocatedCnt++;
      validAllocCnt++;
    }
  }
  // write averaged infos to file:
  for(int i=minArrUsed; i<maxArrUsed; i++){
    int cnt = arrAllocRequestCount[i];
    if(cnt != 0){
      fprintf(fp, "%d %d %lu %lu %lu %lu %lu\n", i, cnt, chunkAllocatedSum[i]/cnt,
            chunkRequestSum[i]/cnt, clockTimingSum[i][0]/cnt,
            clockTimingSum[i][1] / cnt, clockTimingSum[i][2] /cnt);
    } else {
      fprintf(fp, "%d 0 0 0 0 0 0\n", i);
    }        
  }
  // free memory of GPU
  for(int i=0; i<arrCnt; i++){
    if(arrAllocated[i] != 0){
      gpuFree(i, gridSize, blockSize);
    }
  }
  CHECK_CUDA(hipFree(clockTicks));
  CHECK_CUDA(hipFree(allocatedChunksCount));
  CHECK_CUDA_KERNEL(cuNewFreeArrayPointer<<<1,1>>>());
}

int main(int argc, char *argv[]){
  int chunkSize = 4096;
  int maxChunkCountPerAlloc = 1;
  int warmUpRounds = 200; // maybe increase
  int testRounds = 500;
  bool useMallocMC = false;

  if(argc == 1){
    printf("Usage: useMC chunkSize maxChunkSizePerAlloc\n");
    return 0;
  }
  if(argc > 1){
    useMallocMC = (atoi(argv[1]) != 0);
  }
  if(argc > 2){
    chunkSize = atoi(argv[2]);
  }
  if(argc > 3){
    maxChunkCountPerAlloc = atoi(argv[3]);
  }

  srand (time(NULL));
  int freeMemBytes = gpuInit(useMallocMC);
  printf("Useable Memory: %d Bytes\n", freeMemBytes);
  int maxChunks = freeMemBytes / (sizeof(char) *chunkSize );
  int maxThreads = maxChunks / (arrCnt * maxChunkCountPerAlloc); 
  char fname[255];
  if(useMallocMC){
    sprintf(fname, "mallocMC(%d)_%d_%d.txt", MC_CONFIG_ID, chunkSize, maxChunkCountPerAlloc);
  }else{
   sprintf(fname, "new_%d_%d.txt", chunkSize, maxChunkCountPerAlloc);
  }
  const int sizeCnt = 44;
  int gs [sizeCnt] = {1  , 1  , 1  , 1  , 1  , 1  , 1  , 1  , 
                         1  , 1  , 1  , 1  , 1  , 1  , 1  , 1  ,
                         1  , 1  , 1  , 1  , 4  , 5  , 6  , 7  ,
                         8  , 10 , 12 , 14 , 16 , 20 , 24 , 28 ,
                         32 , 40 , 48 , 56 , 64 , 80 , 96 , 128,
                         160, 192, 224, 256};
  int bs [sizeCnt] = {2  , 4  , 6  , 8  , 12 , 16 , 20 , 24 ,
                         28 , 32 , 40 , 48 , 56 , 64 , 80 , 96 ,
                         128, 160, 192, 224, 64 , 64 , 64 , 64 ,
                         64 , 64 , 64 , 64 , 64 , 64 , 64 , 64 ,
                         64 , 64 , 64 , 64 , 64 , 64 , 64 , 64 ,
                         64 , 64 , 64 , 64 };

  
  FILE *fp = fopen(fname, "w");
  // write header of file
  fprintf(fp, "# Contains multiple averaged testruns\n");
  fprintf(fp, "# Test starts with: chunkSize, maxChunkCountPerAlloc,\n");
  fprintf(fp, "#                   useMC, warmUpRounds, testRounds, gridSize,\n");
  fprintf(fp, "#                   blockSize, arrCnt, minArrUsed, maxArrUsed\n");
  fprintf(fp, "# Run lines:  arrUsed, cnt (over all cases), avg alloc mem,\n");
  fprintf(fp, "#             avg min, avg mean, avg max\n");


  for(int i=0; i<sizeCnt; i++){
    int threadCount = gs[i] * bs[i];
    if(threadCount < maxThreads){
      printf("%d ", threadCount);
      fflush(stdout);
      run(chunkSize, maxChunkCountPerAlloc, warmUpRounds, testRounds, 
          gs[i], bs[i], mmc, fp);
    }
  }
  printf("\n");
  fclose(fp); 
  if(mmc != NULL){
    delete mmc;
  }
  hipDeviceReset();
  return 0; 
}

